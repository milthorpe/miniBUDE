#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}
extern "C" {
int get_device_driver_version(const int device) {
    hipSetDevice(device);
    check_error();
    int driver;
    hipDriverGetVersion(&driver);
    check_error();
    return driver;
}

char* get_device_name(const int device) {
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  check_error();
  std::string device_name(props.name);
  char* data = (char*)malloc(sizeof(char)*device_name.length());
  device_name.copy(data, device_name.length());
  return data;
}
}